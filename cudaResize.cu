#include "hip/hip_runtime.h"
#include "cudaResize.h"

// gpuResample
__global__ void gpuResizeSimple(float2 scale, int *input, int iWidth, int *output, int oWidth, int oHeight) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= oWidth || y >= oHeight)
    return;

  const int dx = (x * scale.x);
  const int dy = (y * scale.y);

  const int px = input[dy * iWidth + dx];

  output[y * oWidth + x] = px;
}


__global__ void gpuResizeAverage(float2 scale, int *input, int iWidth, int *output, int oWidth, int oHeight) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= oWidth || y >= oHeight) {
    return;
  }

  int dx = (x * scale.x);
  int dy = (y * scale.y);

  int r = 0, g = 0, b = 0;
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      r += input[(dy + i) * iWidth + (dx + j)] & 0xff;
      g += (input[(dy + i) * iWidth + (dx + j)] >> 8) & 0xff;
      b += (input[(dy + i) * iWidth + (dx + j)] >> 16) & 0xff;
    }
  }

  output[y * oWidth + x] = (r / 9) + ((g / 9) << 8) + ((b / 9) << 16);
}

__global__ void gpuResizeBilinear(float2 scale, int *input, int iWidth, int *output, int oWidth, int oHeight) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= oWidth || y >= oHeight) {
    return;
  }

  int dx = (x * scale.x);
  int dy = (y * scale.y);
  double center_x = dx + scale.x / 2;
  double center_y = dy + scale.y / 2;
  int window[4][3];
  window[0][0] = input[(int) (center_y - scale.y / 2) * iWidth + (int) (center_x - scale.x / 2)] & 0xff;
  window[0][1] = (input[(int) (center_y - scale.y / 2) * iWidth + (int) (center_x - scale.x / 2)] >> 8) & 0xff;
  window[0][2] = (input[(int) (center_y - scale.y / 2) * iWidth + (int) (center_x - scale.x / 2)] >> 16) & 0xff;

  window[1][0] = input[(int) (center_y + scale.y / 2) * iWidth + (int) (center_x - scale.x / 2)] & 0xff;
  window[1][1] = (input[(int) (center_y + scale.y / 2) * iWidth + (int) (center_x - scale.x / 2)] >> 8) & 0xff;
  window[1][2] = (input[(int) (center_y + scale.y / 2) * iWidth + (int) (center_x - scale.x / 2)] >> 16) & 0xff;

  window[2][0] = input[(int) (center_y - scale.y / 2) * iWidth + (int) (center_x + scale.x / 2)] & 0xff;
  window[2][1] = (input[(int) (center_y - scale.y / 2) * iWidth + (int) (center_x + scale.x / 2)] >> 8) & 0xff;
  window[2][2] = (input[(int) (center_y - scale.y / 2) * iWidth + (int) (center_x + scale.x / 2)] >> 16) & 0xff;

  window[3][0] = input[(int) (center_y + scale.y / 2) * iWidth + (int) (center_x + scale.x / 2)] & 0xff;
  window[3][1] = (input[(int) (center_y + scale.y / 2) * iWidth + (int) (center_x + scale.x / 2)] >> 8) & 0xff;
  window[3][2] = (input[(int) (center_y + scale.y / 2) * iWidth + (int) (center_x + scale.x / 2)] >> 16) & 0xff;

  int finalBytes[3];
  for (int i = 0; i < 3; ++i) {
    double x_axis_interpolation_lower = ((int) (center_x + scale.x / 2) - center_x) * window[0][i] /
                                        ((int) (center_x + scale.x / 2) - (int) (center_x - scale.x / 2)) +
                                        (center_x - (int) (center_x - scale.x / 2)) * window[2][i] /
                                        ((int) (center_x + scale.x / 2) - (int) (center_x - scale.x / 2));
    double x_axis_interpolation_higher = ((int) (center_x + scale.x / 2) - center_x) * window[1][i] /
                                         (int(center_x + scale.x / 2) - (int) (center_x - scale.x / 2)) +
                                         (center_x - (int) (center_x - scale.x / 2)) * window[3][i] /
                                         ((int) (center_x + scale.x / 2) - (int) (center_x - scale.x / 2));

    finalBytes[i] =
            ((int) (center_y + scale.y / 2) - center_y) * x_axis_interpolation_lower /
            ((int) (center_y + scale.y / 2) - (int) (center_y - scale.y / 2)) +
            (center_y - (int) (center_y - scale.y / 2)) * x_axis_interpolation_higher /
            ((int) (center_y + scale.y / 2) - (int) (center_y - scale.y / 2));
  }

  output[y * oWidth + x] = finalBytes[0] + (finalBytes[1] << 8) + (finalBytes[2] << 16);
}

// cudaResize
void cudaResize(int *input, size_t inputWidth, size_t inputHeight,
                int *output, size_t outputWidth, size_t outputHeight) {
  if (!input || !output)
    return;

  if (inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0)
    return;

  const float2 scale = make_float2(float(inputWidth) / float(outputWidth),
                                   float(inputHeight) / float(outputHeight));

  // launch kernel
  const dim3 blockDim(8, 8);
//  const dim3 gridDim(outputWidth / blockDim.x, outputHeight / blockDim.y);
  const dim3 gridDim((outputWidth - 1 + blockDim.x - 1) / blockDim.x,
                     (outputHeight - 1 + (blockDim.x - 1)) / blockDim.y);

  int *cudaNewPic, *cudaArrayPic;
  hipMalloc((void **) &cudaNewPic, outputWidth * outputHeight * sizeof(int));
  hipMalloc((void **) &cudaArrayPic, inputWidth * inputHeight * sizeof(int));

  hipMemcpy(cudaArrayPic, input, inputWidth * inputHeight * sizeof(int), hipMemcpyHostToDevice);

  //gpuResize<<< gridDim, blockDim >>>(scale, cudaArrayPic, inputWidth, cudaNewPic, outputWidth, outputHeight);
  //gpuResizeAverage<<< gridDim, blockDim >>>(scale, cudaArrayPic, inputWidth, cudaNewPic, outputWidth, outputHeight);
  gpuResizeBilinear<<< gridDim, blockDim >>>(scale, cudaArrayPic, inputWidth, cudaNewPic, outputWidth, outputHeight);

  hipMemcpy(output, cudaNewPic, outputWidth * outputHeight * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(cudaNewPic);
  hipFree(cudaArrayPic);
}
